#include "hip/hip_runtime.h"
/*
* Copyright 2017-2018 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/

#include <stdint.h>
#include <hip/hip_runtime.h>
#include "../Utils/NvCodecUtils.h"

#define SLEEP_TIME 0

inline __device__ double sleep(int n) {
    double d = 1.0;
    for (int i = 0; i < n; i++) {
        d += sin(d);
    }
    return d;
}

static __global__ void Ripple(uint8_t *pImage, int nWidth, int nHeight, int xCenter, int yCenter, int iTime) {
    int ix = blockIdx.x * blockDim.x + threadIdx.x,
        iy = blockIdx.y * blockDim.y + threadIdx.y;
    if (ix >= nWidth || iy >= nHeight) {
        return;
    }
    float dx = ix - xCenter, dy = iy - yCenter, d = sqrtf(dx * dx + dy * dy), dmax = sqrtf(nWidth * nWidth + nHeight * nHeight) / 2.0f;
    pImage[iy * nWidth + ix] = (uint8_t)(127.0f * (1.0f - d / dmax) * sinf((d - iTime * 10)* 0.1) + 128.0f);
    sleep(SLEEP_TIME);
}

void LaunchRipple(hipStream_t stream, uint8_t *dpImage, int nWidth, int nHeight, int xCenter, int yCenter, int iTime) {
    Ripple<<<dim3((nWidth + 15) / 16, (nHeight + 15) / 16), dim3(16, 16), 0, stream>>>(dpImage, nWidth, nHeight, xCenter, yCenter, iTime);
}

inline __device__ uint8_t clamp(int i) {
    return (uint8_t)min(max(i, 0), 255);
}

static __global__ void OverlayRipple(uint8_t *pNv12, uint8_t *pRipple, int nWidth, int nHeight) {
    int ix = blockIdx.x * blockDim.x + threadIdx.x,
        iy = blockIdx.y * blockDim.y + threadIdx.y;
    if (ix >= nWidth || iy >= nHeight) {
        return;
    }
    pNv12[iy * nWidth + ix] = clamp(pNv12[iy * nWidth + ix] + (pRipple[iy * nWidth + ix] - 127.0f) * 0.8f);
    sleep(SLEEP_TIME);
}

void LaunchOverlayRipple(hipStream_t stream, uint8_t *dpNv12, uint8_t *dpRipple, int nWidth, int nHeight) {
    OverlayRipple<<<dim3((nWidth + 15) / 16, (nHeight + 15) / 16), dim3(16, 16), 0, stream>>>(dpNv12, dpRipple, nWidth, nHeight);
}

static __global__ void Merge(uint8_t *pNv12Merged, uint8_t **apNv12, int nImage, int nWidth, int nHeight) {
    int ix = blockIdx.x * blockDim.x + threadIdx.x,
        iy = blockIdx.y * blockDim.y + threadIdx.y;
    if (ix >= nWidth / 2 || iy >= nHeight / 2) {
        return;
    }
    uint2 y01 = {}, y23 = {}, uv = {};
    for (int i = 0; i < nImage; i++) {
        uchar2 c2;
        c2 = *(uchar2 *)(apNv12[i] + nWidth * iy * 2 + ix * 2);
        y01.x += c2.x; y01.y += c2.y;
        c2 = *(uchar2 *)(apNv12[i] + nWidth * (iy * 2 + 1) + ix * 2);
        y23.x += c2.x; y23.y += c2.y;
        c2 = *(uchar2 *)(apNv12[i] + nWidth * (nHeight + iy) + ix * 2);
        uv.x += c2.x; uv.y += c2.y;
    }
    *(uchar2 *)(pNv12Merged + nWidth * iy * 2 + ix * 2) = uchar2 {(uint8_t)(y01.x / nImage), (uint8_t)(y01.y / nImage)};
    *(uchar2 *)(pNv12Merged + nWidth * (iy * 2 + 1) + ix * 2) = uchar2 {(uint8_t)(y23.x / nImage), (uint8_t)(y23.y / nImage)};
    *(uchar2 *)(pNv12Merged + nWidth * (nHeight + iy) + ix * 2) = uchar2 {(uint8_t)(uv.x / nImage), (uint8_t)(uv.y / nImage)};
    sleep(SLEEP_TIME);
}

void LaunchMerge(hipStream_t stream, uint8_t *dpNv12Merged, uint8_t **pdpNv12, int nImage, int nWidth, int nHeight) {
    uint8_t **dadpNv12;
    ck(hipMalloc(&dadpNv12, sizeof(uint8_t *) * nImage));
    ck(hipMemcpy(dadpNv12, pdpNv12, sizeof(uint8_t *) * nImage, hipMemcpyHostToDevice));
    Merge<<<dim3((nWidth + 15) / 16, (nHeight + 15) / 16), dim3(8, 8), 0, stream>>>(dpNv12Merged, dadpNv12, nImage, nWidth, nHeight);
    ck(hipFree(dadpNv12));
}
