#include "hip/hip_runtime.h"

// BlockDim = 32x16
//GridDim = w/32*h/16
extern "C" __global__ void InterleaveUV( unsigned char *yuv_cb, unsigned char *yuv_cr, unsigned char *nv12_chroma,
                  int chroma_width, int chroma_height, int cb_pitch, int cr_pitch, int nv12_pitch )
{
    int x,y;
    unsigned char *pCb;
    unsigned char *pCr;
    unsigned char *pDst;
    x = blockIdx.x*blockDim.x+threadIdx.x;
    y = blockIdx.y*blockDim.y+threadIdx.y;

    if ((x < chroma_width) && (y < chroma_height))
    {
        pCb = yuv_cb + (y*cb_pitch);
        pCr = yuv_cr + (y*cr_pitch);
        pDst = nv12_chroma + y*nv12_pitch;
        pDst[x << 1]       = pCb[x];
        pDst[(x << 1) + 1] = pCr[x];
    }
}

// Simple NV12 bi-linear scaling using 2D textures
//
// blockDim {64,1,1}

texture<unsigned char, 2> luma_tex;
texture<uchar2, 2>  chroma_tex;

typedef struct {
    uchar2 uv0;
    uchar2 uv1;
} uvpair_t;

extern "C" __global__ void Scale_Bilinear_NV12(unsigned char *dst, int dst_uv_offset,
    int dst_width, int dst_height, int dst_pitch,
    float left, float right,
    float x_offset, float y_offset, float xc_offset, float yc_offset, float x_scale, float y_scale)
{
    unsigned char *dsty, *dstuv;
    uchar4 tmp0, tmp1;
    uvpair_t tmp2;
    int y0, tx;
    float x, yt, yb, yc, leftuv, rightuv;

    tx = (blockIdx.x << 8) + threadIdx.x * 4;
    if (tx < dst_width)
    {
        y0 = blockIdx.y << 1;
        // Luma
        dsty = dst + __umul24(y0, dst_pitch);
        yt = y_offset + (y0 + 0) * y_scale;
        yb = y_offset + (y0 + 1) * y_scale;
        x = 0.5f + fminf(fmaxf(x_offset + (tx + 0) * x_scale, left), right);
        tmp0.x = tex2D(luma_tex, x, yt);
        tmp1.x = tex2D(luma_tex, x, yb);
        x = 0.5f + fminf(fmaxf(x_offset + (tx + 1) * x_scale, left), right);
        tmp0.y = tex2D(luma_tex, x, yt);
        tmp1.y = tex2D(luma_tex, x, yb);
        x = 0.5f + fminf(fmaxf(x_offset + (tx + 2) * x_scale, left), right);
        tmp0.z = tex2D(luma_tex, x, yt);
        tmp1.z = tex2D(luma_tex, x, yb);
        x = 0.5f + fminf(fmaxf(x_offset + (tx + 3) * x_scale, left), right);
        tmp0.w = tex2D(luma_tex, x, yt);
        tmp1.w = tex2D(luma_tex, x, yb);
        *(uchar4 *)(dsty + tx) = tmp0;
        *(uchar4 *)(dsty + tx + dst_pitch) = tmp1;
        // Chroma
		dstuv = dst + dst_uv_offset + __umul24(blockIdx.y, dst_pitch);
        leftuv = 0.5f + 0.5f*left;
        rightuv = 0.5f*(right + 1.0f - left) - 1.0f;
        yc = yc_offset + (y0 >> 1) * y_scale;
        x = leftuv + fminf(fmaxf(xc_offset + (tx >> 1) * x_scale - left, 0.0f), rightuv);
        tmp2.uv0 = tex2D(chroma_tex, x, yc);
        x = leftuv + fminf(fmaxf(xc_offset + ((tx + 2) >> 1) * x_scale - left, 0.0f), rightuv);
        tmp2.uv1 = tex2D(chroma_tex, x, yc);
        *(uvpair_t *)(dstuv + tx) = tmp2;
    }
}
